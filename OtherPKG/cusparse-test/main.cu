#include <msg.h>
#include <common.h>
#include <mmio_highlevel.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <timer.h>

#include <map>

double test_spmv(int m, int n, int nnz, int*row_ptr, int*col_ptr, double*value, double*convert_time)
{
    int* d_row_ptr, *d_col_ptr;
    double* d_value, *d_x, *d_y;

    hipMalloc(&d_row_ptr, (m + 1) * sizeof(int));
    hipMalloc(&d_col_ptr, nnz * sizeof(int));
    hipMalloc(&d_value, nnz * sizeof(double));

    hipMemcpy(d_row_ptr, row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ptr, col_ptr, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_value, value, nnz * sizeof(double), hipMemcpyHostToDevice);
    
    double*x = (double*)malloc(n * sizeof(double));
    #pragma omp parallel for
    for (int i = 0; i < n; ++i)
        x[i] = 1.0;

    hipMalloc(&d_x, n * sizeof(MatValue));
    hipMalloc(&d_y, m * sizeof(MatValue));

    hipMemcpy(d_x, x, n * sizeof(MatValue), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, m * sizeof(MatValue));
    hipFree(x);

    hipsparseSpMatDescr_t matA;
    Timer t;
    timer_start(t);
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseCreateCsr(&matA, m, n, nnz, d_row_ptr, d_col_ptr, d_value, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    timer_end(t);
    echo(debug, "hipsparseCreateCsr: %lf ms", timer_duration(t));
    if (convert_time) {
        *convert_time = timer_duration(t);
    }
    hipsparseDnVecDescr_t vecX, vecY;
    hipsparseCreateDnVec(&vecX, n, d_x, HIP_R_64F);
    hipsparseCreateDnVec(&vecY, m, d_y, HIP_R_64F);

    void *dBuffer1 = NULL, *dBuffer2 = NULL;
    size_t bufferSize1 = 0;
    MatValue alpha = 1.0, beta = 0.0;

    hipsparseStatus_t status1 = hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_64F, (hipsparseSpMVAlg_t)0, &bufferSize1);

    if (status1 != HIPSPARSE_STATUS_SUCCESS) {
        echo(error, "STEP1: %s", hipsparseGetErrorString(status1));
        return -1;
    }

    hipMalloc(&dBuffer1, bufferSize1);

    hipsparseStatus_t status2 = hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_64F, (hipsparseSpMVAlg_t)0, &bufferSize1);

    if (status2 != HIPSPARSE_STATUS_SUCCESS) {
        echo(error, "STEP2: %s", hipsparseGetErrorString(status2));
        return -1;
    }
    
    hipsparseStatus_t status3;
    status3 = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_64F, (hipsparseSpMVAlg_t)0, dBuffer1);
    if (status3 != HIPSPARSE_STATUS_SUCCESS) {
        echo(error, "STEP3: %s", hipsparseGetErrorString(status3));
        return -1;
    }
    
    double first = 0.0; 
    hipMemcpy(&first, d_y, sizeof(double), hipMemcpyDeviceToHost);
    echo(debug, "first: %lf", first);

    // time measurement
    // Timer t;
    timer_start(t);
    for (int i = 0; i < 1000; ++i)
        status3 = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_64F, (hipsparseSpMVAlg_t)0, dBuffer1);
    hipDeviceSynchronize();
    timer_end(t);
    double usingTime = timer_duration(t) / 1000;

    hipFree(d_row_ptr);
    hipFree(d_col_ptr);
    hipFree(d_value);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(dBuffer1);
    hipFree(dBuffer2);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(handle);

    return 2.0 * nnz / usingTime / 1e6;
}

double test_spmv_bsr(int m, int n, int nnz, int*row_ptr, int*col_ptr, double*value)
{
    int* d_row_ptr, *d_col_ptr;
    double* d_value, *d_x, *d_y;

    hipMalloc(&d_row_ptr, (m + 1) * sizeof(int));
    hipMalloc(&d_col_ptr, nnz * sizeof(int));
    hipMalloc(&d_value, nnz * sizeof(double));

    hipMemcpy(d_row_ptr, row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ptr, col_ptr, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_value, value, nnz * sizeof(double), hipMemcpyHostToDevice);
    // convert csr to bsr use cusparseXcsr2bsr
    int blockDim = 16;
    int base;
    int nnzb;
    int *bsrRowPtrC, *bsrColIndC;
    double *bsrValC;
    // create handle
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    // create matrix descriptor
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    // create bsr matrix descriptor
    hipsparseMatDescr_t descrC;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_COLUMN;
    int mb = (m + blockDim-1)/blockDim;
    int nb = (n + blockDim-1)/blockDim;
    hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1));
    hipsparseXcsr2bsrNnz(handle, dirA, m, n,
            descrA, d_row_ptr, d_col_ptr, blockDim,
            descrC, bsrRowPtrC, &nnzb);
    hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb);
    hipMalloc((void**)&bsrValC, sizeof(double)*(blockDim*blockDim)*nnzb);
    hipsparseDcsr2bsr(handle, dirA, m, n,
            descrA, d_value, d_row_ptr, d_col_ptr, blockDim,
            descrC, bsrValC, bsrRowPtrC, bsrColIndC);
    // step 2: allocate vector x and vector y large enough for bsrmv
    double *hx = (double*)malloc(n * sizeof(double));
    #pragma omp parallel for
    for (int i = 0; i < n; ++i)
        hx[i] = 1.0;
    
    hipMalloc(&d_x, sizeof(double)*n);
    hipMalloc(&d_y, sizeof(double)*(mb*blockDim));
    hipMemcpy(d_x, hx, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemset(d_y, 0, sizeof(double)*(mb*blockDim));
    free(hx);
    MatValue alpha = 1.0, beta = 0.0;
    // step 3: perform bsrmv
    hipsparseDbsrmv(handle, dirA, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha,
    descrC, bsrValC, bsrRowPtrC, bsrColIndC, blockDim, d_x, &beta, d_y);
    
    // perform time measurement
    Timer t;
    timer_start(t);
    for (int i = 0; i < 1000; ++i)
        hipsparseDbsrmv(handle, dirA, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha,
        descrC, bsrValC, bsrRowPtrC, bsrColIndC, blockDim, d_x, &beta, d_y);
    hipDeviceSynchronize();
    timer_end(t);
    double usingTime = timer_duration(t) / 1000;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_row_ptr);
    hipFree(d_col_ptr);
    hipFree(d_value);
    hipFree(bsrRowPtrC);
    hipFree(bsrColIndC);
    hipFree(bsrValC);
    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyMatDescr(descrC);
    return 2.0 * nnz / usingTime / 1e6;
}

double test_spmm(int m, int n, int nnz, int*row_ptr, int*col_ptr, double*value, int right_n)
{
    // A is a m x n sparse matrix, B is a n x right_n matrix, C is a m x right_n matrix
    hipsparseHandle_t handle = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipsparseSpMatDescr_t A = NULL;
    hipsparseDnMatDescr_t B = NULL;
    hipsparseDnMatDescr_t C = NULL;
    void *dBuffer = NULL;
    size_t bufferSize = 0;

    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    
    MatIndex *d_csrRowPtr, *d_csrColIdx;
    MatValue *d_csrVal, *d_b, *d_c;
    hipMalloc((void **)&d_csrRowPtr, (m + 1) * sizeof(MatIndex));
    hipMalloc((void **)&d_csrColIdx, nnz * sizeof(MatIndex));
    hipMalloc((void **)&d_csrVal, nnz * sizeof(MatValue));
    hipMalloc((void **)&d_b, n * right_n * sizeof(MatValue));
    hipMalloc((void **)&d_c, m * right_n * sizeof(MatValue));

    MatValue *b = (MatValue *)malloc(n * right_n * sizeof(MatValue));
    #pragma omp parallel for
    for (int i = 0; i < n * right_n; ++i)
        b[i] = 1.0;

    hipMemcpy(d_csrRowPtr, row_ptr, (m + 1) * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIdx, col_ptr, nnz * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal, value, nnz * sizeof(MatValue), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * right_n * sizeof(MatValue), hipMemcpyHostToDevice);
    hipMemset(d_c, 0, m * right_n * sizeof(MatValue));

    hipsparseCreateCsr(&A, m, n, nnz, d_csrRowPtr, d_csrColIdx, d_csrVal, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateDnMat(&B, n, right_n, n, d_b, HIP_R_64F, HIPSPARSE_ORDER_COL);
    hipsparseCreateDnMat(&C, m, right_n, m, d_c, HIP_R_64F, HIPSPARSE_ORDER_COL);

    MatValue alpha = 1.0, beta = 0.0;
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A, B, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);

    hipMalloc(&dBuffer, bufferSize);
    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < 1000; ++i)
    {
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A, B, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float duration;
    hipEventElapsedTime(&duration, start, stop);
    duration /= 1000;

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(A);
    hipsparseDestroyDnMat(B);
    hipsparseDestroyDnMat(C);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(handle);
    hipFree(dBuffer);
    hipFree(d_csrRowPtr);
    hipFree(d_csrColIdx);
    hipFree(d_csrVal);
    hipFree(d_b);
    hipFree(d_c);
    return 2.0 * nnz * right_n / duration / 1e6;
}

double test_spmm_coo(int m, int n, int nnz, int*row_ptr, int*col_ptr, double*value, int right_n)
{
    // A is a m x k sparse matrix, B is a k x n matrix, C is a m x n matrix

    int*row_idx = (int *)malloc(nnz * sizeof(int));
    #pragma omp parallel for
    for (int i = 0; i < m; i++)
    {
        for (int j = row_ptr[i]; j < row_ptr[i + 1]; j++) row_idx[j] = i;
    }

    hipsparseHandle_t handle = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipsparseSpMatDescr_t A = NULL;
    hipsparseDnMatDescr_t B = NULL;
    hipsparseDnMatDescr_t C = NULL;
    void *dBuffer = NULL;
    size_t bufferSize = 0;

    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    
    MatIndex *d_csrRowPtr, *d_csrColIdx;
    MatValue *d_csrVal, *d_b, *d_c;
    hipMalloc((void **)&d_csrRowPtr, nnz * sizeof(MatIndex));
    hipMalloc((void **)&d_csrColIdx, nnz * sizeof(MatIndex));
    hipMalloc((void **)&d_csrVal, nnz * sizeof(MatValue));
    hipMalloc((void **)&d_b, n * right_n * sizeof(MatValue));
    hipMalloc((void **)&d_c, m * right_n * sizeof(MatValue));

    MatValue *b = (MatValue *)malloc(n * right_n * sizeof(MatValue));
    #pragma omp parallel for
    for (int i = 0; i < n * right_n; ++i)
        b[i] = 1.0;

    hipMemcpy(d_csrRowPtr, row_idx, nnz * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIdx, col_ptr, nnz * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal, value, nnz * sizeof(MatValue), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * right_n * sizeof(MatValue), hipMemcpyHostToDevice);
    hipMemset(d_c, 0, m * right_n * sizeof(MatValue));

    hipsparseCreateCoo(&A, m, n, nnz, d_csrRowPtr, d_csrColIdx, d_csrVal, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateDnMat(&B, n, right_n, right_n, d_b, HIP_R_64F, HIPSPARSE_ORDER_ROW);
    hipsparseCreateDnMat(&C, m, right_n, right_n, d_c, HIP_R_64F, HIPSPARSE_ORDER_ROW);

    MatValue alpha = 1.0, beta = 0.0;
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A, B, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);

    hipMalloc(&dBuffer, bufferSize);
    Timer timer;
    timer_start(timer);
    for (int i = 0; i < 1000; ++i)
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A, B, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);
    hipDeviceSynchronize();
    timer_end(timer);
    double duration = timer_duration(timer) / 1000;

    // destroy matrix/vector descriptors
    free(row_idx);
    hipsparseDestroySpMat(A);
    hipsparseDestroyDnMat(B);
    hipsparseDestroyDnMat(C);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(handle);
    hipFree(dBuffer);
    hipFree(d_csrRowPtr);
    hipFree(d_csrColIdx);
    hipFree(d_csrVal);
    hipFree(d_b);
    hipFree(d_c);
    return 2.0 * nnz * right_n / duration / 1e6;
}

double test_spgemm(int m, int n, int nnz, int*row_ptr, int*col_ptr, double*value)
{
    MatIndex *d_row_ptr, *d_col_ptr;
    MatValue *d_value, alpha = 1.0, beta = 0.0;

    uint64_t intermidiate = 0;
    #pragma omp parallel for reduction(+:intermidiate)
    for (int i = 0; i < m; ++i)
    {
        uint64_t sum = 0;
        for (int j = row_ptr[i]; j < row_ptr[i + 1]; ++j)
        {
            int col = col_ptr[j];
            sum += row_ptr[col + 1] - row_ptr[col];
        }
        intermidiate += sum;
    }

    hipMalloc(&d_row_ptr, (m + 1) * sizeof(MatIndex));
    hipMalloc(&d_col_ptr, nnz * sizeof(MatIndex));
    hipMalloc(&d_value, nnz * sizeof(MatValue));

    hipMemcpy(d_row_ptr, row_ptr, (m + 1) * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ptr, col_ptr, nnz * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_value, value, nnz * sizeof(MatValue), hipMemcpyHostToDevice);

    double duration = 0;
    int repeat = 10;
    for (int i = 0; i < repeat; ++i) {
        hipsparseHandle_t handle;
        hipsparseCreate(&handle);

        hipsparseSpMatDescr_t matA, matB, matC;
        hipsparseCreateCsr(&matA, m, n, nnz, d_row_ptr, d_col_ptr, d_value, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
        hipsparseCreateCsr(&matB, m, n, nnz, d_row_ptr, d_col_ptr, d_value, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
        hipsparseCreateCsr(&matC, m, n, 0, NULL, NULL, NULL, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

        hipsparseSpGEMMDescr_t spgemmDesc;
        hipsparseSpGEMM_createDescr(&spgemmDesc);

        size_t buffer_size1 = 0, buffer_size2 = 0;
        void *buffer1 = NULL, *buffer2 = NULL;
        
        Timer t1, t2, t3, t4;
        timer_start(t1);
        auto status = hipsparseSpGEMM_workEstimation(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size1, NULL);
        timer_end(t1);
        
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            echo(error, "STEP1: %s", hipsparseGetErrorString(status));
            return -1;
        }

        hipMalloc(&buffer1, buffer_size1);
        timer_start(t2);
        status = hipsparseSpGEMM_workEstimation(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size1, buffer1);
        timer_end(t2);
        
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            echo(error, "STEP2: %s", hipsparseGetErrorString(status));
            return -1;
        }

        timer_start(t3);
        status = hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size2, NULL);
        timer_end(t3);
        
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            echo(error, "STEP3: %s", hipsparseGetErrorString(status));
            return -1;
        } 

        hipMalloc(&buffer2, buffer_size2);

        timer_start(t4);
        hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size2, buffer2);
        hipsparseSpGEMM_copy(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);
        timer_end(t4);
        double used_time = timer_duration(t1) + timer_duration(t2) + timer_duration(t3) + timer_duration(t4);
        // echo(debug, "duration: %lf ms, step1: %lf ms, step2: %lf ms, step3: %lf ms, step4: %lf ms", used_time, timer_duration(t1), timer_duration(t2), timer_duration(t3), timer_duration(t4));
        duration += used_time;

        int64_t Cm, Cn, Cnnz;
        hipsparseSpMatGetSize(matC, &Cm, &Cn, &Cnnz);

        if (Cnnz == 0) {
            echo(error, "Cnnz == 0");
            return -1;
        }

        hipsparseDestroy(handle);
        hipsparseDestroySpMat(matA);
        hipsparseDestroySpMat(matB);
        hipsparseDestroySpMat(matC);
        hipsparseSpGEMM_destroyDescr(spgemmDesc);
        hipFree(buffer1);
        hipFree(buffer2);
    }
    duration /= repeat;
    hipFree(d_row_ptr);
    hipFree(d_col_ptr);
    hipFree(d_value);
    echo(debug, "intermidiate: %lu, duration: %lf", intermidiate, duration);
    return intermidiate * 2.0 / duration / 1e6;
}

double test_spgemm_coo(int m, int n, int nnz, int*row_ptr, int*col_ptr, double*value)
{

    int*row_idx = (int *)malloc(nnz * sizeof(int));
    #pragma omp parallel for
    for (int i = 0; i < m; i++)
    {
        for (int j = row_ptr[i]; j < row_ptr[i + 1]; j++) row_idx[j] = i;
    }

    MatIndex *d_row_ptr, *d_col_ptr;
    MatValue *d_value, alpha = 1.0, beta = 0.0;

    uint64_t intermidiate = 0;
    #pragma omp parallel for reduction(+:intermidiate)
    for (int i = 0; i < m; ++i)
    {
        uint64_t sum = 0;
        for (int j = row_ptr[i]; j < row_ptr[i + 1]; ++j)
        {
            int col = col_ptr[j];
            sum += row_ptr[col + 1] - row_ptr[col];
        }
        intermidiate += sum;
    }

    hipMalloc(&d_row_ptr, nnz * sizeof(MatIndex));
    hipMalloc(&d_col_ptr, nnz * sizeof(MatIndex));
    hipMalloc(&d_value, nnz * sizeof(MatValue));

    hipMemcpy(d_row_ptr, row_idx, nnz * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ptr, col_ptr, nnz * sizeof(MatIndex), hipMemcpyHostToDevice);
    hipMemcpy(d_value, value, nnz * sizeof(MatValue), hipMemcpyHostToDevice);

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipsparseSpMatDescr_t matA, matB, matC;
    hipsparseCreateCoo(&matA, m, n, nnz, d_row_ptr, d_col_ptr, d_value, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateCoo(&matB, m, n, nnz, d_row_ptr, d_col_ptr, d_value, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateCoo(&matC, m, n, 0, NULL, NULL, NULL, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseSpGEMMDescr_t spgemmDesc;
    hipsparseSpGEMM_createDescr(&spgemmDesc);

    size_t buffer_size1 = 0, buffer_size2 = 0;
    void *buffer1 = NULL, *buffer2 = NULL;

    hipsparseSpGEMM_workEstimation(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size1, NULL);

    hipMalloc(&buffer1, buffer_size1);
    hipsparseSpGEMM_workEstimation(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size1, buffer1);

    hipsparseSpGEMM_compute(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size2, NULL);
    hipMalloc(&buffer2, buffer_size2);
    Timer t;
    timer_start(t);
    if (
        HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES == hipsparseSpGEMM_compute(
                                                      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                      &alpha, matA, matB, &beta, matC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffer_size2, buffer2))
    {
        echo(error, "insufficient resources\n");
    }
    timer_end(t);

    hipsparseDestroy(handle);
    hipsparseDestroySpMat(matA);
    hipsparseDestroySpMat(matB);
    hipsparseDestroySpMat(matC);
    hipsparseSpGEMM_destroyDescr(spgemmDesc);

    hipFree(d_row_ptr);
    hipFree(d_col_ptr);
    hipFree(d_value);
    hipFree(buffer1);
    hipFree(buffer2);
    return intermidiate * 2.0 / timer_duration(t) / 1e6;
}

int main(int argc, char*argv[])
{
    int m, n, nnz, is_symmetric;
    int *row_ptr, *col_ptr;
    double *value;
    int device_id = atoi(argv[argc-2]);
    hipSetDevice(device_id);
    // print device info
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    echo(info, "Device %d: %s, compute capability: %d.%d", device_id, prop.name, prop.major, prop.minor);

    mmio_allinone(argv[1], &m, &n, &nnz, &is_symmetric, &row_ptr, &col_ptr, &value);
    std::string op = argv[argc - 1];
    double gflops, convert_time = 0;

    if (op == "--spmv")
    {
        gflops = test_spmv(m, n, nnz, row_ptr, col_ptr, value, &convert_time);
    } else if (op == "--spmm")
    {
        int right_n = atoi(argv[argc-3]);
        gflops = test_spmm(m, n, nnz, row_ptr, col_ptr, value, right_n);
    } else if (op == "--spgemm")
    {
        gflops = test_spgemm(m, n, nnz, row_ptr, col_ptr, value);
    } else if (op == "--spmm-coo")
    {
        int right_n = atoi(argv[argc-3]);
        gflops = test_spmm_coo(m, n, nnz, row_ptr, col_ptr, value, right_n);
    } else if (op == "--spgemm-coo")
    {
        gflops = test_spgemm_coo(m, n, nnz, row_ptr, col_ptr, value);
    } else
    {
        echo(error, "Invalid operation: \"%s\"", op.c_str());
    }

    printf("%.3lf,%.3lf\n", convert_time, gflops);

    free(row_ptr);
    free(col_ptr);
    free(value);
    return 0;
}

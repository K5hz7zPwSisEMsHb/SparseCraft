#include <tmatrix/DataStructure/hd.cuh>

BaseMatrix* BaseMatrix_Host_to_Device(BaseMatrix*mm)
{
    BaseMatrix *dmm_host = (BaseMatrix *)malloc(sizeof(BaseMatrix));

    MatIndex* tile_row_ptr;
    MatIndex* tile_col_idx;
    Tile* tiles;
    char* data;

    hipError_t e = hipMalloc(&tile_row_ptr, (mm->_m + 1) * sizeof(MatIndex)); 
    if (e != hipSuccess || tile_row_ptr == NULL) {
        echo(error, "hipMalloc tile_row_ptr failed\n");
    }
    e = hipMalloc(&tile_col_idx, mm->_nnz * sizeof(MatIndex));
    if (e != hipSuccess || tile_col_idx == NULL) {
        echo(error, "hipMalloc tile_col_idx failed\n");
    }
    e = hipMalloc(&tiles, mm->_nnz * sizeof(Tile));
    if (e != hipSuccess || tiles == NULL) {
        echo(error, "hipMalloc tiles failed\n");
    }
    e = hipMalloc(&data, mm->_data_len);
    if (e != hipSuccess || data == NULL) {
        echo(error, "hipMalloc data failed\n");
    }

    e = hipMemcpy(tile_row_ptr, mm->tile_row_ptr, (mm->_m + 1) * sizeof(MatIndex), hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_row_ptr failed\n");
    }
    e = hipMemcpy(tile_col_idx, mm->tile_col_idx, mm->_nnz * sizeof(MatIndex),     hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_col_idx failed\n");
    }
    e = hipMemcpy(tiles,        mm->tiles,        mm->_nnz * sizeof(Tile),         hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tiles failed\n");
    }
    e = hipMemcpy(data,         mm->data,         mm->_data_len,                   hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy data failed\n");
    }

    dmm_host->meta_m = mm->meta_m;
    dmm_host->meta_n = mm->meta_n;
    dmm_host->meta_nnz = mm->meta_nnz;
    dmm_host->_m = mm->_m;
    dmm_host->_n = mm->_n;
    dmm_host->_nnz = mm->_nnz;
    dmm_host->tile_row_ptr = tile_row_ptr;
    dmm_host->tile_col_idx = tile_col_idx;
    dmm_host->tiles        = tiles;
    dmm_host->data         = data;

    return dmm_host;
}

BaseMatrixCSC* BaseMatrix_Host_to_Device(BaseMatrixCSC*mm)
{
    BaseMatrixCSC *dmm_host = (BaseMatrixCSC *)malloc(sizeof(BaseMatrixCSC));

    MatIndex* tile_col_ptr;
    MatIndex* tile_row_idx;
    Tile* tiles;
    char* data;

    hipError_t e = hipMalloc(&tile_col_ptr, (mm->_m + 1) * sizeof(MatIndex)); 
    if (e != hipSuccess || tile_col_ptr == NULL) {
        echo(error, "hipMalloc tile_col_ptr failed\n");
    }
    e = hipMalloc(&tile_row_idx, mm->_nnz * sizeof(MatIndex));
    if (e != hipSuccess || tile_row_idx == NULL) {
        echo(error, "hipMalloc tile_row_idx failed\n");
    }
    e = hipMalloc(&tiles, mm->_nnz * sizeof(Tile));
    if (e != hipSuccess || tiles == NULL) {
        echo(error, "hipMalloc tiles failed\n");
    }
    e = hipMalloc(&data, mm->_data_len);
    if (e != hipSuccess || data == NULL) {
        echo(error, "hipMalloc data failed\n");
    }

    e = hipMemcpy(tile_col_ptr, mm->tile_col_ptr, (mm->_m + 1) * sizeof(MatIndex), hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_col_ptr failed\n");
    }
    e = hipMemcpy(tile_row_idx, mm->tile_row_idx, mm->_nnz * sizeof(MatIndex),     hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_row_idx failed\n");
    }
    e = hipMemcpy(tiles,        mm->tiles,        mm->_nnz * sizeof(Tile),         hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tiles failed\n");
    }
    e = hipMemcpy(data,         mm->data,         mm->_data_len,                   hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy data failed\n");
    }

    dmm_host->meta_m = mm->meta_m;
    dmm_host->meta_n = mm->meta_n;
    dmm_host->meta_nnz = mm->meta_nnz;
    dmm_host->_m = mm->_m;
    dmm_host->_n = mm->_n;
    dmm_host->_nnz = mm->_nnz;
    dmm_host->tile_col_ptr = tile_col_ptr;
    dmm_host->tile_row_idx = tile_row_idx;
    dmm_host->tiles        = tiles;
    dmm_host->data         = data;

    return dmm_host;
}

BaseMatrix*BaseMatrix_Device_to_Host(BaseMatrix*mm)
{
    BaseMatrix *hmm_device = (BaseMatrix *)malloc(sizeof(BaseMatrix));

    MatIndex* tile_row_ptr = (MatIndex *)malloc((mm->_m + 1) * sizeof(MatIndex));
    MatIndex* tile_col_idx = (MatIndex *)malloc(mm->_nnz * sizeof(MatIndex));
    Tile* tiles = (Tile *)malloc(mm->_nnz * sizeof(Tile));
    char* data = (char *)malloc(mm->_data_len);

    hipError_t e = hipMemcpy(tile_row_ptr, mm->tile_row_ptr, (mm->_m + 1) * sizeof(MatIndex), hipMemcpyDeviceToHost);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_row_ptr failed\n");
    }
    e = hipMemcpy(tile_col_idx, mm->tile_col_idx, mm->_nnz * sizeof(MatIndex),     hipMemcpyDeviceToHost);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_col_idx failed\n");
    }
    e = hipMemcpy(tiles,        mm->tiles,        mm->_nnz * sizeof(Tile),         hipMemcpyDeviceToHost);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tiles failed\n");
    }
    e = hipMemcpy(data,         mm->data,         mm->_data_len,                   hipMemcpyDeviceToHost);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy data failed\n");
    }

    hmm_device->meta_m = mm->meta_m;
    hmm_device->meta_n = mm->meta_n;
    hmm_device->meta_nnz = mm->meta_nnz;
    hmm_device->_m = mm->_m;
    hmm_device->_n = mm->_n;
    hmm_device->_nnz = mm->_nnz;
    hmm_device->tile_row_ptr = tile_row_ptr;
    hmm_device->tile_col_idx = tile_col_idx;
    hmm_device->tiles        = tiles;
    hmm_device->data         = data;

    return hmm_device;
}

void DestroyBaseMatrixHost(BaseMatrix *hmm)
{
    free(hmm->tile_row_ptr);
    free(hmm->tile_col_idx);
    free(hmm->tiles);
    // free(hmm->data);
    free(hmm);
}

void DestroyBaseMatrixHost(BaseMatrixCSC *hmm)
{
    free(hmm->tile_col_ptr);
    free(hmm->tile_row_idx);
    free(hmm->tiles);
    // free(hmm->data);
    free(hmm);
}

void DestroyBaseMatrixDevice(BaseMatrix *dmm)
{
    hipFree(dmm->tile_row_ptr);
    hipFree(dmm->tile_col_idx);
    hipFree(dmm->tiles);
    hipFree(dmm->data);
    free(dmm);
}

void DestroyBaseMatrixDevice(BaseMatrixCSC *dmm)
{
    hipFree(dmm->tile_col_ptr);
    hipFree(dmm->tile_row_idx);
    hipFree(dmm->tiles);
    hipFree(dmm->data);
    free(dmm);
}
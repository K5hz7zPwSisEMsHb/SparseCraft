#include <tmatrix/common.h>
#include <hip/hip_runtime.h>
#include <tmatrix/Utils/msg.h>

bool file_exists(const char* filename)
{
    FILE*f = fopen(filename, "r");
    if (f == NULL)
        return false;
    fclose(f);
    return true;
}

void cudaInit(int device)
{
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    echo(info, "Device %d: %s, compute capability: %d.%d", device, prop.name, prop.major, prop.minor);
}

void cudaDebug(std::string&msg)
{
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        echo(error, "%s: %s", msg.c_str(), hipGetErrorString(err));
        exit(1);
    }
}
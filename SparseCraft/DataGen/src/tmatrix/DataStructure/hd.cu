#include <tmatrix/DataStructure/hd.cuh>

BaseMatrix* BaseMatrix_Host_to_Device(BaseMatrix*mm)
{
    BaseMatrix *dmm_host = (BaseMatrix *)malloc(sizeof(BaseMatrix));

    MatIndex* tile_row_ptr;
    MatIndex* tile_col_idx;
    Tile* tiles;
    char* data;

    hipError_t e = hipMalloc(&tile_row_ptr, (mm->_m + 1) * sizeof(MatIndex)); 
    if (e != hipSuccess || tile_row_ptr == NULL) {
        echo(error, "hipMalloc tile_row_ptr failed\n");
    }
    e = hipMalloc(&tile_col_idx, mm->_nnz * sizeof(MatIndex));
    if (e != hipSuccess || tile_col_idx == NULL) {
        echo(error, "hipMalloc tile_col_idx failed\n");
    }
    e = hipMalloc(&tiles, mm->_nnz * sizeof(Tile));
    if (e != hipSuccess || tiles == NULL) {
        echo(error, "hipMalloc tiles failed\n");
    }
    e = hipMalloc(&data, mm->_data_len);
    if (e != hipSuccess || data == NULL) {
        echo(error, "hipMalloc data failed\n");
    }

    e = hipMemcpy(tile_row_ptr, mm->tile_row_ptr, (mm->_m + 1) * sizeof(MatIndex), hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_row_ptr failed\n");
    }
    e = hipMemcpy(tile_col_idx, mm->tile_col_idx, mm->_nnz * sizeof(MatIndex),     hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tile_col_idx failed\n");
    }
    e = hipMemcpy(tiles,        mm->tiles,        mm->_nnz * sizeof(Tile),         hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy tiles failed\n");
    }
    e = hipMemcpy(data,         mm->data,         mm->_data_len,                   hipMemcpyHostToDevice);
    if (e != hipSuccess) {
        echo(error, "hipMemcpy data failed\n");
    }

    dmm_host->meta_m = mm->meta_m;
    dmm_host->meta_n = mm->meta_n;
    dmm_host->meta_nnz = mm->meta_nnz;
    dmm_host->_m = mm->_m;
    dmm_host->_n = mm->_n;
    dmm_host->_nnz = mm->_nnz;
    dmm_host->tile_row_ptr = tile_row_ptr;
    dmm_host->tile_col_idx = tile_col_idx;
    dmm_host->tiles        = tiles;
    dmm_host->data         = data;

    return dmm_host;
}

void DestroyBaseMatrixHost(BaseMatrix *hmm)
{
    free(hmm->tile_row_ptr);
    free(hmm->tile_col_idx);
    free(hmm->tiles);
    // free(hmm->data);
    free(hmm);
}

void DestroyBaseMatrixDevice(BaseMatrix *dmm)
{
    hipFree(dmm->tile_row_ptr);
    hipFree(dmm->tile_col_idx);
    hipFree(dmm->tiles);
    hipFree(dmm->data);
    free(dmm);
}